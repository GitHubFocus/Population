#include "hip/hip_runtime.h"
#include "popconfig.h"

#include <iostream>

#if defined(HAVE_CUDA)
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#endif

#include "c_neural_net.h"
#include "popcuda.h"
#include "Population.h"
#include "microtime.h"

struct layer {
	unsigned int _X_size; // size of _X and _d_E_X
	unsigned int _Y_size; // size of _Y and _d_E_Y
	unsigned int _W_width; // width of _W and _d_E_W
	unsigned int _W_height; // height of _W and _d_E_W
	bool _errors_initialized;
	pop::F32* _X;
	pop::F32* _Y;
	pop::F32* _W;
	pop::F32* _d_E_X;
	pop::F32* _d_E_Y;
	pop::F32* _d_E_W;
};

struct neural_network {
	double _eta;
	unsigned int _nb_layers;
	struct layer* _layers;
};

const int EPOCH=500;

GPUNeuralNetwork::GPUNeuralNetwork(std::vector<unsigned int> v_layer, double eta) {
	createNetwork(v_layer, eta);
#if defined(HAVE_CUDA)
	copyNetworkToGPU();
#endif
}

GPUNeuralNetwork::~GPUNeuralNetwork() {
	deleteNetwork();
#if defined(HAVE_CUDA)
	deleteNetworkOnGPU();
#endif
}


void GPUNeuralNetwork::createNetwork(std::vector<unsigned int> v_layer, double eta) {
	h_network = new struct neural_network;

	h_network->_nb_layers = v_layer.size();
	h_network->_layers = new struct layer[h_network->_nb_layers];
	h_network->_eta = eta;

	for(unsigned int i=0;i<v_layer.size();i++){
		int size_layer = v_layer[i];
		struct layer& l = h_network->_layers[i];

		if(i != v_layer.size()-1) {
			// add a bias neuron with constant value 1
			l._X_size = size_layer+1;
		} else {
			// except for the last layer
			l._X_size = size_layer;
		}
		l._X = new pop::F32[l._X_size];
		for (unsigned int j=0; j<l._X_size; j++) {
			l._X[j] = 1;
		}
		l._d_E_X = NULL;

		l._Y_size = size_layer;
		l._Y = new pop::F32[l._Y_size];
		for (unsigned int j=0; j<l._Y_size; j++) {
			l._Y[j] = 0;
		}
		l._d_E_Y = NULL;

		if (i != 0) {
			unsigned int size_layer_previous = h_network->_layers[i-1]._X_size;
			pop::DistributionNormal n(0,1./std::sqrt(size_layer_previous));

			l._W_height = size_layer;
			l._W_width = size_layer_previous;
			l._W = new pop::F32[l._W_height * l._W_width];
			for (unsigned int j=0; j<l._W_height * l._W_width; j++) {
				l._W[j] = n.randomVariable();
			}
		} else {
			l._W_height = 0;
			l._W_width = 0;
			l._W = NULL;
		}
		l._d_E_W = NULL;
	}
}

void GPUNeuralNetwork::deleteNetwork() {
	for (unsigned int i=0; i<h_network->_nb_layers; i++) {
		struct layer& l = h_network->_layers[i];

		delete[] l._X;
		if (l._d_E_X != NULL) {
			delete[] l._d_E_X;
		}

		delete[] l._Y;
		if (l._d_E_Y != NULL) {
			delete[] l._d_E_Y;
		}

		if (l._W != NULL) {
			delete[] l._W;
		}
		if (l._d_E_W != NULL) {
			delete[] l._d_E_W;
		}
	}
	delete[] h_network->_layers;
	delete h_network;
}


void GPUNeuralNetwork::printNeuronsVector(pop::F32* V, unsigned int size, std::string label) {
	if (V == NULL) {
		std::cout << label << " = NULL" << std::endl;
	} else {
		std::cout << label << "(" << size << ") = [";
		for (unsigned int i=0; i<size; i++) {
			std::cout << "\t" << V[i];
		}
		std::cout << "\t]" << std::endl;
	}
}

void GPUNeuralNetwork::printWeightMatrix(pop::F32* M, unsigned int height, unsigned int width, std::string label) {
	if (M == NULL) {
		std::cout << label << " = NULL" << std::endl;
	} else {
		std::cout << label << "(" << height << ", " << width << ") = [" << std::endl;
		for (unsigned int i=0; i<height; i++) {
			for (unsigned int j=0; j<width; j++) {
				std::cout << "\t" << M[i*width + j];
			}
			std::cout << std::endl;
		}
		std::cout << "]" << std::endl;
	}
}

void GPUNeuralNetwork::displayNetwork() {
	std::cout << "Number of layers: " << h_network->_nb_layers << ", eta: " << h_network->_eta << std::endl;

	for (unsigned int l=0; l<h_network->_nb_layers; l++) {
		struct layer& layer = h_network->_layers[l];

		std::cout << "\n-- Layer " << l << ", _X_size = " << layer._X_size << ", Y_size = " << layer._Y_size << ", _W_height = " << layer._W_height << ", _W_width = " << layer._W_width << std::endl;
		printNeuronsVector(layer._X, layer._X_size, "_X");
		printNeuronsVector(layer._Y, layer._Y_size, "_Y");
		printWeightMatrix(layer._W, layer._W_height, layer._W_width, "_W");
		printNeuronsVector(layer._d_E_X, layer._X_size, "_d_E_X");
		printNeuronsVector(layer._d_E_Y, layer._Y_size, "_d_E_Y");
		printWeightMatrix(layer._d_E_W, layer._W_height, layer._W_width, "_d_E_W");
	}
}

void GPUNeuralNetwork::setEta(const double eta) {
	h_network->_eta = eta;
}

double GPUNeuralNetwork::getEta() const {
	return h_network->_eta;
}

void GPUNeuralNetwork::propagateFront(const pop::VecF32& in , pop::VecF32 &out) {
	std::copy(in.begin(),in.end(), h_network->_layers[0]._X);

	for (unsigned int l=0; l<h_network->_nb_layers-1; l++) {
		struct layer& prev_layer = h_network->_layers[l];
		struct layer& layer = h_network->_layers[l+1];

		// _Y[l+1] = _W[l+1] * _X[l]
		for (unsigned int i=0; i<layer._Y_size; i++) {
			layer._Y[i] = 0;
			for (unsigned int j=0; j<prev_layer._X_size; j++) {
				layer._Y[i] += layer._W[i*prev_layer._X_size+j] * prev_layer._X[j];
			}
		}

		// _X[l+1] = sigmoid(_Y[l+1])
		for (unsigned int i=0; i<layer._Y_size; i++) {
			layer._X[i] = sigmoid(layer._Y[i]);
		}
	}

	struct layer& last_layer = h_network->_layers[h_network->_nb_layers-1];
	if (out.size() != last_layer._X_size) {
		out.resize(last_layer._X_size);
	}
	std::copy(last_layer._X, last_layer._X+last_layer._X_size,out.begin());
}

void GPUNeuralNetwork::propagateBackFirstDerivate(const pop::VecF32& desired_output) {
	for (unsigned int l=0; l<h_network->_nb_layers; l++) {
		struct layer& layer = h_network->_layers[l];
		if (layer._d_E_X == NULL) {
			layer._d_E_X = new pop::F32[layer._X_size];
			memcpy(layer._d_E_X, layer._X, sizeof(layer._X[0]) * layer._X_size);
		}
		if (layer._d_E_Y == NULL) {
			layer._d_E_Y = new pop::F32[layer._Y_size];
			memcpy(layer._d_E_Y, layer._Y, sizeof(layer._X[0]) * layer._Y_size);
		}
		if (layer._W != NULL && layer._d_E_W == NULL) {
			layer._d_E_W = new pop::F32[layer._W_height*layer._W_width];
			memcpy(layer._d_E_W, layer._W, sizeof(layer._W[0]) * layer._W_height*layer._W_width);
		}
	}

	for (unsigned int l=h_network->_nb_layers-1; l>0; l--) {
		struct layer& layer = h_network->_layers[l];
		struct layer& prev_layer = h_network->_layers[l-1];

		// _d_E_X[l] = _X[l] - desired_output
		if (l == h_network->_nb_layers-1){
			for (unsigned int j=0; j<layer._X_size; j++) {
				layer._d_E_X[j] = layer._X[j] - desired_output[j];
			}
		}

		// _d_E_Y[l] = _d_E_X[l] * derived_sigmoid(_X[l])
		for (unsigned int j=0; j<layer._Y_size; j++) {
			layer._d_E_Y[j] = layer._d_E_X[j] * derived_sigmoid(layer._X[j]);
		}

		// _d_E_W[l-1] = _d_E_Y[l] * _X[l-1]
		// _W[l-1] = _W[l-1] - _eta * _d_E_W[l-1]
		for(unsigned int j=0; j<layer._W_width; j++){
			for (unsigned int i=0; i<layer._W_height; i++) {
				int idx = i*layer._W_width+j;
				layer._d_E_W[idx] = layer._d_E_Y[i] * prev_layer._X[j];
				layer._W[idx] = layer._W[idx] - h_network->_eta*layer._d_E_W[idx];
			}
		}

		// _d_E_X[l-1][j] = sum_{i=0}^{_W[l-1].sizeI()}{_W[l](i, j) * _d_E_Y[l](i)}, j=0 to _X[l].size()
		for(unsigned int j=0; j<prev_layer._X_size; j++){
			prev_layer._d_E_X[j] = 0;
			for (unsigned int i=0; i<layer._W_height; i++) {
				prev_layer._d_E_X[j] += layer._W[i*layer._W_width+j] * layer._d_E_Y[i];
			}
		}
	}
}


#if defined(HAVE_CUDA)
static const char* cublasGetErrorString(hipblasStatus_t status)
{
	switch(status)
	{
	case HIPBLAS_STATUS_SUCCESS: return "HIPBLAS_STATUS_SUCCESS";
	case HIPBLAS_STATUS_NOT_INITIALIZED: return "HIPBLAS_STATUS_NOT_INITIALIZED";
	case HIPBLAS_STATUS_ALLOC_FAILED: return "HIPBLAS_STATUS_ALLOC_FAILED";
	case HIPBLAS_STATUS_INVALID_VALUE: return "HIPBLAS_STATUS_INVALID_VALUE";
	case HIPBLAS_STATUS_ARCH_MISMATCH: return "HIPBLAS_STATUS_ARCH_MISMATCH";
	case HIPBLAS_STATUS_MAPPING_ERROR: return "HIPBLAS_STATUS_MAPPING_ERROR";
	case HIPBLAS_STATUS_EXECUTION_FAILED: return "HIPBLAS_STATUS_EXECUTION_FAILED";
	case HIPBLAS_STATUS_INTERNAL_ERROR: return "HIPBLAS_STATUS_INTERNAL_ERROR";
	}
	return "unknown error";
}

void GPUNeuralNetwork::copyNetworkToGPU() {
	// * in-memory representation on the gpu *
	// We allocate a big continuous array that will contain all the structures + values
	// [struct neural_network | struct layer 1 | struct layer 2 | ... | struct layer n | *_X | *_Y | *_W | *_d_E_X | *_d_E_Y | *_d_E_W |	 		  ...			 ]
	//																				   [  				for layer 1					   ][ for layer 2 ] [ for others ]

	unsigned int size = sizeof(*h_network) + h_network->_nb_layers * sizeof(h_network->_layers[0]);
	for (unsigned int i=0; i<h_network->_nb_layers; i++) {
		struct layer& layer = h_network->_layers[i];
		size += (layer._X_size + layer._Y_size) * 2 * sizeof(layer._X[0]);
		if (i!=0) {
			size += (layer._W_height + layer._W_width) * 2 * sizeof(layer._W[0]);
		}
	}
	hipMalloc(&d_network, size);

	struct layer* p_layers =  h_network->_layers;
	h_network->_layers = (struct layer*)(d_network+1);
	hipMemcpy(d_network, h_network, sizeof(*h_network), hipMemcpyHostToDevice);
	h_network->_layers = p_layers;

	p_layers = (struct layer*)(d_network+1);
	pop::F32* start = (pop::F32*)((char*)d_network + sizeof(*d_network) + h_network->_nb_layers * sizeof(*p_layers));
	for (unsigned int i=0; i<h_network->_nb_layers; i++) {
		struct layer& layer = h_network->_layers[i];

		pop::F32* p_X = layer._X;
		pop::F32* p_Y = layer._Y;
		pop::F32* p_W = layer._W;
		pop::F32* p_d_E_X = layer._d_E_X;
		pop::F32* p_d_E_Y = layer._d_E_Y;
		pop::F32* p_d_E_W = layer._d_E_W;

		layer._X = start;
		layer._Y = layer._X + layer._X_size;
		layer._W = layer._Y + layer._Y_size;

		layer._d_E_X = layer._W + layer._W_height*layer._W_width;
		layer._d_E_Y = layer._d_E_X + layer._X_size;
		layer._d_E_W = layer._d_E_Y + layer._Y_size;

		// Note: we do not need to copy the errors vectors (i.e., d_E_*), as they will be initialized during the propagateBack algorithm
		hipMemcpy(layer._X, p_X, sizeof(*p_X) * layer._X_size, hipMemcpyHostToDevice);
		hipMemcpy(layer._Y, p_Y, sizeof(*p_Y) * layer._Y_size, hipMemcpyHostToDevice);
		if (i!=0) {
			hipMemcpy(layer._W, p_W, sizeof(*p_W) * layer._W_height*layer._W_width, hipMemcpyHostToDevice);
		} else {
			layer._W = NULL;
		}
		hipMemcpy(p_layers, &layer, sizeof(*p_layers), hipMemcpyHostToDevice);

		start = layer._d_E_W + layer._W_height*layer._W_width;

		layer._X = p_X;
		layer._Y = p_Y;
		layer._W = p_W;
		layer._d_E_X = p_d_E_X;
		layer._d_E_Y = p_d_E_Y;
		layer._d_E_W = p_d_E_W;

		p_layers++;
	}
}

void GPUNeuralNetwork::copyNetworkFromGPU() {
	hipMemcpy(h_network, d_network, sizeof(*h_network), hipMemcpyDeviceToHost);

	struct layer* p_layers =  h_network->_layers;
	h_network->_layers = new struct layer[h_network->_nb_layers];

	pop::F32* start = (pop::F32*)((char*)d_network + sizeof(*d_network) + h_network->_nb_layers * sizeof(*p_layers));
	for (unsigned int i=0; i<h_network->_nb_layers; i++) {
		struct layer& layer = h_network->_layers[i];

		hipMemcpy(&layer, &p_layers[i], sizeof(*p_layers), hipMemcpyDeviceToHost);

		layer._X = new pop::F32[layer._X_size];
		hipMemcpy(layer._X, start, sizeof(layer._X[0])*layer._X_size, hipMemcpyDeviceToHost);
		start += layer._X_size;

		layer._Y = new pop::F32[layer._Y_size];
		hipMemcpy(layer._Y, start, sizeof(layer._Y[0])*layer._Y_size, hipMemcpyDeviceToHost);
		start += layer._Y_size;

		if (i!=0) {
			layer._W = new pop::F32[layer._W_height * layer._W_width];
			hipMemcpy(layer._W, start, sizeof(layer._W[0])*layer._W_height*layer._W_width, hipMemcpyDeviceToHost);
			start += layer._W_height * layer._W_width;
		} else {
			layer._W_height = 0;
			layer._W_width = 0;
			layer._W = NULL;
		}

		// We do not need to copy the errors vectors (i.e., d_E_*), as they will be initialized during the propagateBack algorithm
		layer._d_E_X = NULL;
		layer._d_E_Y = NULL;
		layer._d_E_W = NULL;

		start += layer._X_size + layer._Y_size + layer._W_height*layer._W_width;
	}
}

void GPUNeuralNetwork::deleteNetworkOnGPU() {
	hipFree(d_network);
}

__device__ void printVectorOnGPU(pop::F32* V, unsigned int size, char* label) {
	printf("%s = [", label);
	for (unsigned int i=0; i<size; i++) {
		printf(" %f", V[i]);
	}
	printf("]\n");
}

__device__ void printMatrixOnGPU(pop::F32* M, unsigned int height, unsigned int width, char* label) {
	printf("%s = [\n", label);
	for (unsigned int i=0; i<height; i++) {
		for (unsigned int j=0; j<width; j++) {
			printf(" %f", M[i*width + j]);
		}
		printf("\n");
	}
	printf("]\n");
}

__global__ void printNetworkOnGPU(struct neural_network *network) {
	printf("Number of layers: %d, eta: %f\n", network->_nb_layers, network->_eta);
	for (unsigned int l=0; l<network->_nb_layers; l++) {
		struct layer& layer = network->_layers[l];
		printf("\n--Layer %d, _X_size = %d, _Y_size = %d, _W_height = %d, _W_width = %d\n", l, layer._X_size, layer._Y_size, layer._W_height, layer._W_width);

		printVectorOnGPU(layer._X, layer._X_size, (char*)"_X");
		printVectorOnGPU(layer._Y, layer._Y_size, (char*)"_Y");
		printMatrixOnGPU(layer._W, layer._W_height, layer._W_width, (char*)"_W");
		printVectorOnGPU(layer._d_E_X, layer._X_size, (char*)"_d_E_X");
		printVectorOnGPU(layer._d_E_Y, layer._Y_size, (char*)"_d_E_Y");
		printMatrixOnGPU(layer._d_E_W, layer._W_height, layer._W_width, (char*)"_d_E_W");
	}
}

void GPUNeuralNetwork::gpu_displayNetwork() {
	printNetworkOnGPU<<<1, 1>>>(d_network);
	hipDeviceSynchronize();
}

__global__ void gpu_propagateFront_setInput(struct neural_network *network, pop::F32* in_set, unsigned int in_elt_size, unsigned int idx) {
	int tid = blockDim.x*blockIdx.x + threadIdx.x;
	if (tid < in_elt_size) {
		network->_layers[0]._X[tid] = in_set[idx*in_elt_size+tid];
	}
}

__global__ void gpu_propagateFront_computeSigmoid(struct neural_network *network, int l) {
	int tid = blockDim.x*blockIdx.x + threadIdx.x;
	if (tid < network->_layers[l]._Y_size) {
		network->_layers[l]._X[tid] = 1.7159f*tanhf(0.66666667f*network->_layers[l]._Y[tid]);
	}
}

__global__ void gpu_propagateFront_setOutput(struct neural_network *network, pop::F32* out_computed) {
	int tid = blockDim.x*blockIdx.x + threadIdx.x;
	if (tid < network->_layers[network->_nb_layers-1]._X_size) {
		out_computed[tid] = network->_layers[network->_nb_layers-1]._X[tid];
	}
}

/*
 * Propagate in_set[idx] in out_computed (they must reside in GPU memory)
 * in_set: set of all the inputs. Each element's size is in_elt_size
 * out_computed: the output element, of size equal to the number of neurons in the last layer
 */
void GPUNeuralNetwork::gpu_propagateFront(pop::F32* in_set, unsigned int in_elt_size, unsigned int idx, pop::F32* out_computed) {
	int block, grid;

	block = (h_network->_layers[0]._X_size < MAX_NB_THREADS ? h_network->_layers[0]._X_size : MAX_NB_THREADS);
	grid = h_network->_layers[0]._X_size / MAX_NB_THREADS + (h_network->_layers[0]._X_size%MAX_NB_THREADS ? 1 : 0);
	gpu_propagateFront_setInput<<<grid, block>>>(d_network, in_set, in_elt_size, idx);
	hipDeviceSynchronize();

	hipblasStatus_t	stat;
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	float alpha = 1.0f;
	float beta = 0.0f;
	pop::F32* start = (pop::F32*)((char*)d_network + sizeof(*d_network) + h_network->_nb_layers * sizeof(h_network->_layers[0]));

	for (unsigned int l=0; l<h_network->_nb_layers-1; l++) {
		struct layer& prev_layer = h_network->_layers[l];
		struct layer& layer = h_network->_layers[l+1];

		pop::F32* d_X = start;
		start += (prev_layer._X_size + prev_layer._Y_size + prev_layer._W_height*prev_layer._W_width)*2; // d_Y and d_W are from the next layer
		pop::F32* d_Y = start + layer._X_size;
		pop::F32* d_W = d_Y + layer._Y_size;

		// _Y[l+1] = _W[l+1] * _X[l]
		stat = hipblasSgemv(handle, HIPBLAS_OP_T, layer._W_width, layer._W_height, &alpha, d_W, prev_layer._X_size, d_X, 1, &beta, d_Y, 1);
		if (stat != HIPBLAS_STATUS_SUCCESS) {
			std::cout << "Cublas error in _Y[l+1] = _W[l+1] * _X[l] for layer l = " << l << ", cublas status: " << cublasGetErrorString(stat) << std::endl;
		}

		// _X[l+1] = sigmoid(_Y[l+1])
		block = (h_network->_layers[l+1]._X_size < MAX_NB_THREADS ? h_network->_layers[l+1]._X_size : MAX_NB_THREADS);
		grid = h_network->_layers[l+1]._X_size / MAX_NB_THREADS + (h_network->_layers[l+1]._X_size%MAX_NB_THREADS ? 1 : 0);
		gpu_propagateFront_computeSigmoid<<<grid, block>>>(d_network, l+1);
		hipDeviceSynchronize();
	}

	hipblasDestroy(handle);

	block = (h_network->_layers[h_network->_nb_layers-1]._X_size < MAX_NB_THREADS ? h_network->_layers[h_network->_nb_layers-1]._X_size : MAX_NB_THREADS);
	grid = h_network->_layers[h_network->_nb_layers-1]._X_size / MAX_NB_THREADS + (h_network->_layers[h_network->_nb_layers-1]._X_size%MAX_NB_THREADS ? 1 : 0);
	gpu_propagateFront_setOutput<<<grid, block>>>(d_network, out_computed);
	hipDeviceSynchronize();
}

__global__ void gpu_propagateBackFirstDerivate_setXError(struct neural_network *network, pop::F32* desired_output, unsigned int in_elt_size, unsigned int idx, int l) {
	int tid = blockDim.x*blockIdx.x + threadIdx.x;
	if (tid < network->_layers[l]._X_size) {
		network->_layers[l]._d_E_X[tid] = network->_layers[l]._X[tid] - desired_output[idx*in_elt_size+tid];
	}
}

__global__ void gpu_propagateBackFirstDerivate_setYError(struct neural_network *network, int l) {
	int tid = blockDim.x*blockIdx.x + threadIdx.x;
	if (tid < network->_layers[l]._X_size) {
		float S = network->_layers[l]._X[tid];
		network->_layers[l]._d_E_Y[tid] = network->_layers[l]._d_E_X[tid] * (0.666667f/1.7159f*(1.7159f*1.7159f-S*S));
	}
}

__global__ void gpu_propagateBackFirstDerivate_setWeight(struct neural_network *network, int l) {
	int tid = blockDim.x*blockIdx.x + threadIdx.x;
	struct layer& layer = network->_layers[l];
	if (tid < layer._W_height*layer._W_width) {
		int i = tid / layer._W_width;
		int j = tid % layer._W_width;

		//int idx = i*layer._W_width+j;
		//printf("l=%d, i=%d, j=%d, tid=%d, idx=%d\n", l, i, j, tid, idx);

		layer._d_E_W[tid] = layer._d_E_Y[i] * network->_layers[l-1]._X[j];
		layer._W[tid] = layer._W[tid] - network->_eta*layer._d_E_W[tid];
	}
}

__global__ void gpu_propagateBackFirstDerivate_setPreviousXError(struct neural_network *network, int l) {
	int tid = blockDim.x*blockIdx.x + threadIdx.x;
	if (tid < network->_layers[l-1]._X_size) {
		struct layer& layer = network->_layers[l];
		struct layer& prev_layer = network->_layers[l-1];
		pop::F32 s = 0.0f;

		for (unsigned int i=0; i<layer._W_height; i++) {
			s += layer._W[i*layer._W_width+tid] * layer._d_E_Y[i];
		}
		prev_layer._d_E_X[tid] = s;
	}
}

/*
 * Propagate back diff(out_set[idx], out_computed) using the network on the GPU
 * out_set: set of all the inputs. Size = out_set_size. Each element's size is out_elt_size
 * out_computed: the output element computed previously (using propagateFrontGPU), of size out_elt_size
 */
void GPUNeuralNetwork::gpu_propagateBackFirstDerivate(pop::F32* out_set, pop::F32* out_computed, unsigned int out_set_size, unsigned int out_elt_size, unsigned int idx, int* error) {
	int block, grid;

	for (unsigned int l=h_network->_nb_layers-1; l>0; l--) {
		// _d_E_X[l] = _X[l] - desired_output
		if (l == h_network->_nb_layers-1){
			block = (h_network->_layers[l]._X_size < MAX_NB_THREADS ? h_network->_layers[l]._X_size : MAX_NB_THREADS);
			grid = h_network->_layers[l]._X_size / MAX_NB_THREADS + (h_network->_layers[l]._X_size%MAX_NB_THREADS ? 1 : 0);
			gpu_propagateBackFirstDerivate_setXError<<<grid, block>>>(d_network, out_set, out_elt_size, idx, l);
			hipDeviceSynchronize();
		}

		// _d_E_Y[l] = _d_E_X[l] * derived_sigmoid(_X[l])
		block = (h_network->_layers[l]._Y_size < MAX_NB_THREADS ? h_network->_layers[l]._Y_size : MAX_NB_THREADS);
		grid = h_network->_layers[l]._Y_size / MAX_NB_THREADS + (h_network->_layers[l]._Y_size%MAX_NB_THREADS ? 1 : 0);
		gpu_propagateBackFirstDerivate_setYError<<<grid, block>>>(d_network, l);
		hipDeviceSynchronize();

		// _d_E_W[l-1] = _d_E_Y[l] * _X[l-1]
		// _W[l-1] = _W[l-1] - _eta * _d_E_W[l-1]
		unsigned int nb_weights = h_network->_layers[l]._W_height * h_network->_layers[l]._W_width;
		block = (nb_weights < MAX_NB_THREADS ? nb_weights : MAX_NB_THREADS);
		grid = nb_weights / MAX_NB_THREADS + (nb_weights%MAX_NB_THREADS ? 1 : 0);
		gpu_propagateBackFirstDerivate_setWeight<<<grid, block>>>(d_network, l);
		hipDeviceSynchronize();

		// _d_E_X[l-1][j] = sum_{i=0}^{_W[l-1].sizeI()}{_W[l](i, j) * _d_E_Y[l](i)}, j=0 to _X[l].size()
		// _W[l-1] = _W[l-1] - _eta * _d_E_W[l-1]
		block = (h_network->_layers[l-1]._X_size < MAX_NB_THREADS ? h_network->_layers[l-1]._X_size : MAX_NB_THREADS);
		grid = h_network->_layers[l-1]._X_size / MAX_NB_THREADS + (h_network->_layers[l-1]._X_size%MAX_NB_THREADS ? 1 : 0);
		gpu_propagateBackFirstDerivate_setPreviousXError<<<grid, block>>>(d_network, l);
		hipDeviceSynchronize();
	}
}
#endif

void test_neural_net_cpu(void) {
	std::vector<unsigned int> v_layer;
	v_layer.push_back(2);
	v_layer.push_back(3);
	v_layer.push_back(1);
	GPUNeuralNetwork network(v_layer, 0.01);

	std::cout << "\n********** CPU **********\n" << std::endl;

	//create the training set
	// (-1,-1)->-1
	// ( 1,-1)-> 1
	// (-1, 1)-> 1
	// ( 1, 1)->-1
	pop::Vec<pop::VecF32> v_in(4,pop::VecF32(2));//4 vector of two scalar values
	v_in(0)(0)=-1;v_in(0)(1)=-1; // (-1,-1)
	v_in(1)(0)= 1;v_in(1)(1)=-1; // ( 1,-1)
	v_in(2)(0)=-1;v_in(2)(1)= 1; // (-1, 1)
	v_in(3)(0)= 1;v_in(3)(1)= 1; // ( 1, 1)

	pop::Vec<pop::VecF32> v_out(4,pop::VecF32(1));//4 vector of one scalar value
	v_out(0)(0)=-1;// -1
	v_out(1)(0)= 1;//  1
	v_out(2)(0)= 1;//  1
	v_out(3)(0)=-1;// -1

	//use the backpropagation algorithm with first order method
	std::vector<int> v_global_rand(v_in.size());
	for(unsigned int i=0;i<v_global_rand.size();i++)
		v_global_rand[i]=i;
	std::cout<<"iter_epoch\t error_train"<<std::endl;

	unsigned int nbr_epoch = EPOCH;
	for(unsigned int i=0;i<nbr_epoch;i++){
		std::random_shuffle ( v_global_rand.begin(), v_global_rand.end() , pop::Distribution::irand());
		int error=0;
		for(unsigned int j=0;j<v_global_rand.size();j++){
			pop::VecF32 vout;
			network.propagateFront(v_in(v_global_rand[j]),vout);
			network.propagateBackFirstDerivate(v_out(v_global_rand[j]));

			int label1 = std::distance(vout.begin(),std::max_element(vout.begin(),vout.end()));
			int label2 = std::distance(v_out(v_global_rand[j]).begin(),std::max_element(v_out(v_global_rand[j]).begin(),v_out(v_global_rand[j]).end()));
			if(label1!=label2)
				error++;
		}

		//FIXME std::cout<<i<<"\t"<<error*1.0/v_global_rand.size()<<std::endl;
	}

	//test the training
	for(int j=0;j<4;j++){
		pop::VecF32 vout;
		network.propagateFront(v_in(j), vout);
		std::cout<<vout<<std::endl;// we obtain the expected value -1 , 1 , 1 , -1
	}
	std::cout<<std::endl;
}

#if defined(HAVE_CUDA)
void test_neural_net_gpu(void) {
	std::vector<unsigned int> v_layer;
	v_layer.push_back(2);
	v_layer.push_back(3);
	v_layer.push_back(1);
	GPUNeuralNetwork network(v_layer, 0.01);

	std::cout << "\n********** GPU **********\n" << std::endl;

	//create the training set
	pop::Vec<pop::VecF32> v_in(4,pop::VecF32(2));//4 vector of two scalar values
	v_in(0)(0)=-1;v_in(0)(1)=-1; // (-1,-1)
	v_in(1)(0)= 1;v_in(1)(1)=-1; // ( 1,-1)
	v_in(2)(0)=-1;v_in(2)(1)= 1; // (-1, 1)
	v_in(3)(0)= 1;v_in(3)(1)= 1; // ( 1, 1)
	pop::Vec<pop::VecF32> v_out(4,pop::VecF32(1));//4 vector of one scalar value
	v_out(0)(0)=-1;// -1
	v_out(1)(0)= 1;//  1
	v_out(2)(0)= 1;//  1
	v_out(3)(0)=-1;// -1

	size_t total_size_sets = (v_in.size()*v_in(0).size() + v_out.size()*v_out(0).size()) * sizeof(v_in(0)(0));
	size_t free, total;
	hipMemGetInfo(&free, &total);
	if (total_size_sets > .9*free) { // 90% of the free memory
		std::cerr << "Not enough memory on the GPU to process the whole sets at once. You need to copy the sets pieces by pieces" << std::endl;
		return;
	}

	//use the backpropagation algorithm with first order method
	std::vector<int> v_global_rand(v_in.size());
	for(unsigned int i=0;i<v_global_rand.size();i++)
		v_global_rand[i]=i;

	pop::F32* d_in_set;
	hipMalloc(&d_in_set, v_in.size()*v_in(0).size() * sizeof(v_in(0)(0)));
	pop::F32* start = d_in_set;
	for (int i=0; i<v_in.size(); i++) {
		for (int j=0; j<v_in(i).size(); j++) {
			hipMemcpy(start, &v_in(i)(j), sizeof(*d_in_set), hipMemcpyHostToDevice);
			start++;
		}
	}

	pop::F32* d_out_set;
	hipMalloc(&d_out_set, v_out.size()*v_out(0).size() * sizeof(v_in(0)(0)));
	start = d_out_set;
	for (int i=0; i<v_out.size(); i++) {
		for (int j=0; j<v_out(i).size(); j++) {
			hipMemcpy(start, &v_out(i)(j), sizeof(*d_out_set), hipMemcpyHostToDevice);
			start++;
		}
	}

	pop::F32* d_out;
	hipMalloc(&d_out, v_out(0).size() * sizeof(v_in(0)(0)));

	int error;
	int* d_error;
	hipMalloc(&d_error, sizeof(error));

	unsigned int nbr_epoch = EPOCH;
	std::cout<<"iter_epoch\t error_train"<<std::endl;
	for(unsigned int i=0;i<nbr_epoch;i++){
		std::random_shuffle ( v_global_rand.begin(), v_global_rand.end() , pop::Distribution::irand());

		error = 0;
		hipMemcpy(d_error, &error, sizeof(error), hipMemcpyHostToDevice);

		for(unsigned int j=0;j<v_global_rand.size();j++){
			network.gpu_propagateFront(d_in_set, v_in(0).size(), v_global_rand[j], d_out);
			network.gpu_propagateBackFirstDerivate(d_out_set, d_out, v_out.size(), v_out(0).size(), v_global_rand[j], d_error);
			//TODO computeErrorGPU<<<1, 1>>>(d_out_set, d_out, v_out(0).size(), v_global_rand[j], d_error);
		}

		hipMemcpy(&error, d_error, sizeof(error), hipMemcpyDeviceToHost);
		//FIXME std::cout<<i<<"\t"<<error*1.0/v_global_rand.size()<<std::endl;
	}

	hipFree(d_error);
	hipFree(d_out);
	hipFree(d_in_set);
	hipFree(d_out_set);

	network.copyNetworkFromGPU();

	//test the training
	for(int j=0;j<4;j++){
		pop::VecF32 vout;
		network.propagateFront(v_in(j), vout);
		std::cout<<vout<<std::endl;// we obtain the expected value -1 , 1 , 1 , -1
	}
	std::cout<<std::endl;
}

void test_neural_net_gpu_mnist(void) {
	pop::Vec<pop::Vec<pop::Mat2UI8> > number_training =  pop::TrainingNeuralNetwork::loadMNIST("/media/pl/shared/PL/neural_nets_samples/MNIST/train-images-idx3-ubyte","/media/pl/shared/PL/neural_nets_samples/MNIST/train-labels-idx1-ubyte");
	pop::Vec<pop::Vec<pop::Mat2UI8> > number_test =  pop::TrainingNeuralNetwork::loadMNIST("/media/pl/shared/PL/neural_nets_samples/MNIST/t10k-images-idx3-ubyte","/media/pl/shared/PL/neural_nets_samples/MNIST/t10k-labels-idx1-ubyte");

	double size_in= number_training(0)(0).getDomain()(0) * number_training(0)(0).getDomain()(1);
	std::cout << "size trainings: " << number_training(0).size() << std::endl;

	std::vector<unsigned int> v_layer;
	v_layer.push_back(size_in);
	v_layer.push_back(1000);
	v_layer.push_back(1000);
	v_layer.push_back(number_training.size());
	GPUNeuralNetwork network(v_layer, 0.001);

	pop::Vec<pop::VecF32> vtraining_in;
	pop::Vec<pop::VecF32> vtraining_out;

	double ratio = 1;
	pop::TrainingNeuralNetwork::convertMatrixToInputValueNeuron(vtraining_in,vtraining_out,number_training,number_training(0)(0).getDomain(),pop::NNLayerMatrix::Mass,pop::NNLayerMatrix::MinusOneToOne);

	pop::Vec<pop::VecF32> vtest_in;
	pop::Vec<pop::VecF32> vtest_out;
	pop::TrainingNeuralNetwork::convertMatrixToInputValueNeuron(vtest_in,vtest_out,number_test,number_training(0)(0).getDomain(),pop::NNLayerMatrix::Mass,pop::NNLayerMatrix::MinusOneToOne);

	number_training.clear();
	number_test.clear();


	size_t total_size_training = (vtraining_in.size()*vtraining_in(0).size() + vtraining_out.size()*vtraining_out(0).size()) * sizeof(vtraining_in(0)(0));
	size_t total_size_test = (vtest_in.size()*vtest_in(0).size() + vtest_out.size()*vtest_out(0).size()) * sizeof(vtest_in(0)(0));
	std::cout << "total training size: " << total_size_training << ", total size test: " << total_size_test << std::endl;

	std::vector<int> v_global_rand(vtraining_in.size());
	for(unsigned int i=0;i<v_global_rand.size();i++)
		v_global_rand[i]=i;

	std::cout<<"iter_epoch\t error_train\t error_test\t learning rate"<<std::endl;

	for(unsigned int i=0;i<100;i++){
		std::random_shuffle ( v_global_rand.begin(), v_global_rand.end() ,pop::Distribution::irand());
		int error_training=0,error_test=0;

		for(unsigned int j=0;j<v_global_rand.size();j++){
			pop::VecF32 vout;
			network.propagateFront(vtraining_in(v_global_rand[j]),vout);
			int label1 = std::distance(vout.begin(),std::max_element(vout.begin(),vout.end()));
			network.propagateBackFirstDerivate(vtraining_out(v_global_rand[j]));
			int label2 = std::distance(vtraining_out(v_global_rand[j]).begin(),std::max_element(vtraining_out(v_global_rand[j]).begin(),vtraining_out(v_global_rand[j]).end()));
			if(label1!=label2){
				error_training++;
			}
		}
		for(unsigned int j=0;j<vtest_in.size();j++){
			pop::VecF32 vout;
			network.propagateFront(vtest_in(j),vout);
			int label1 = std::distance(vout.begin(),std::max_element(vout.begin(),vout.end()));
			int label2 = std::distance(vtest_out(j).begin(),std::max_element(vtest_out(j).begin(),vtest_out(j).end()));
			if(label1!=label2){
				error_test++;
			}
		}

		network.setEta(network.getEta()*0.9);
		std::cout<<i<<"\t"<<error_training*1./v_global_rand.size()<<"\t"<<error_test*1./vtest_in.size() <<"\t"<<network.getEta()<<std::endl;
	}
}

void test_cublas(void) {
	hipblasStatus_t	stat;
	hipblasHandle_t handle;

	const int width = 3;
	const int height = 2;
	float* W = new float[width*height];
	for (int i=0; i<width*height; i++) {
		W[i] = i;
	}

	std::cout << "W = [";
	for (int i=0; i<width*height; i++) {
		std::cout << " " << W[i];
	}
	std::cout << " ]" << std::endl;

	float* X = new float[width];
	X[0] = 2;
	for (int i=1; i<width; i++) {
		X[i] = 1;
	}

	std::cout << "X = [";
	for (int i=0; i<width; i++) {
		std::cout << " " << X[i];
	}
	std::cout << " ]" << std::endl;

	float* Y = new float[height];
	for (int i=1; i<height; i++) {
		Y[i] = 0;
	}

	float* d_W;
	hipMalloc(&d_W, width*height*sizeof(*d_W));
	hipMemcpy(d_W, W, width*height*sizeof(*d_W), hipMemcpyHostToDevice);

	float* d_X;
	hipMalloc(&d_X, width*sizeof(*d_X));
	hipMemcpy(d_X, X, width*sizeof(*d_X), hipMemcpyHostToDevice);

	float* d_Y;
	hipMalloc(&d_Y, height*sizeof(*d_Y));

	hipblasCreate(&handle);

	float alpha = 1.0f;
	float beta = 0.0f;
	//  Y = α op(W) X + β Y
	stat = hipblasSgemv(handle, HIPBLAS_OP_T, width, height, &alpha, d_W, width, d_X, 1, &beta, d_Y, 1);	// Y = [ 3 15 ]
	std::cout << "cublas status: " << cublasGetErrorString(stat) << std::endl;

	hipMemcpy(Y, d_Y, height*sizeof(*d_Y), hipMemcpyDeviceToHost);

	std::cout << "Y = [";
	for (int i=0; i<height; i++) {
		std::cout << " " << Y[i];
	}
	std::cout << " ]" << std::endl;

	hipblasDestroy(handle);
	hipFree(d_W);
	hipFree(d_X);
	hipFree(d_Y);

	delete[] Y;
	delete[] X;
	delete[] W;
}
#endif
